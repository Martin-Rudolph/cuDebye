#include "hip/hip_runtime.h"
// cuDEBYE SOURCE CODE VERSION 1.5
// TO DO:
// - REWRITE TO DOUBLE PRECISION DISTANCE CALCULATIONS FOR BENCHMARKING
// - CONSIDER NOT CALLING SQRT (HISTOGRAM OF VALUE UNDER SQUARE -> problem with memory, no solution jet) IN KERNEL TO SAVE COMPUTATION TIME
// - USE INTEGER VALUES INSTEAD OF FLOAT AND CALCULATE IN FEMTO METERS INSTEAD OF ANGSTROM -> INTEGER OPERATIONS SHOULD REPLACE ROUND AND SINGLE PRECISION OPERATIONS WITH ACCEPTABLE ERROR
// - IMPLEMENT A CLEVER ALGORYTHM TO SET GRID AND BLOCK SIZE AUTOMATICALLY
// - BINARY FILE SUPPORT FOR FASTER INFORMATION EXCHANGE AND LESS MEMORY CONSUMPTION OR/AND PYTHON7MATLAB INTERFACE TO GET ARRAYS DIRECTLY
// - CREATE INTERFACE TO DISCUS (READ DISCUS STRUCTURES)
// - IMPLEMENT USAGE OF MORE GPU'S
// - MULTIPLE EMPTY LINES IN ASCII CAN CAUSE A CRASH DURING READING
// - HOST AND THRUST OPERATIONS ARE VERY INEFFICIENT (BUT FAST ENOUGH) -> MAYBE REWRITE THEM
// - ELIMINATE COMPILER WARNINGS FOR A MORE STABLE PROGRAM


////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// PREAMBLE: LIBARIES AND USEFULL BASIC FUNCTIONS
////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// Include cuda libaries for parallel computing
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <sm_20_atomic_functions.h>

// Thrust libaries from the cuda toolkit for optimized vector operations
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>
#include <thrust/copy.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>

// Libaries for input and output streams for display results and read and write files.
// Better than the old printf shit
#include <fstream>		// File Stream
#include <iostream>		// Input/Output Stream
#include <iomanip>    // For Output Precision
#include <sstream>		// String Stream -> much easier than char[size] because size is set automatically
using namespace std;	// Normally all stream functions have to called via prefix std:: -> So functions can called withaout prefix (Example: std::cout -> cout)

						// Libary for measuring calculation time
#include <ctime>

						// define the mathematical constant pi
# define PI 3.14159265358979323846

						// Function to check if input file parsed via commandline exists
bool fexists(const char *filename) {
	ifstream file(filename);
	file.close();
	return bool(file.good());
}

// CUDA ERROR CHECK: can be wrapped around every device function, to abort and report if an error occurs.
// I just copied this from an guy of the Stack Overflow Community. 
// http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// ESSENTIAL KERNEL FUNCTIONS FOR FAST DISTANCE CALCULATION AND HISTOGRAM GENERATION
// INCLUDING A CONTROLLING HOST FUNCTION FOR PROPER KERNEL SELECTION
////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// There is no hardware support for floating-point divisions (or integer divisions, for that matter) on the GPU,
// so these operations are implemented as software  subroutines that require additional registers for temporary
// storage.
// This is why 1/r is calculated by the host and passed as rp_rstep to the device and kernels -> saves a lot of computational time
// if thic calculation is avoided in the kernel.
// Additionally it should be noted that: 
// - The less operations within a kernel the faster the kernel is.
// - Single precision operations are faster than double precision operations and simple +,-,* operations are faster then complex math like power, sqrt, etc.
// - If a value is needed twice or more for some further calculation store this value in a local variable and access it to avoid a new calcualtion
//   in normal programs the time do not increase essentially but here this operation is called massivley multiple times which will result in increased
//   computing time.
// - Use pointers for larger arrys to prevent the transgfer of large arrays to the kernel each time it is called. But be aware of the fact that
//   the acess to the global memory is slower than to the local and shared memory of a kernel function.
// ...
// Kernel which will calculate and count all distances withaout checking if the passed atom indices exist.
__global__ void dist2(int *start, float *rp_rstep, float *x, float *y, float *z, unsigned int *nr) {
	// We pretend to have a part of distance matrix with atoms of the first subset in columns and the second subset in
	// the rows. Since all atom positions are stored together in three arrays, the kernel has to calculate the indcices
	// for the atom in row and column of the imaginary distance matrix. Therefore the start value depending on subgroup and
	// subset for the actual grid (see main and splitCalcDist function -- necessary because distance matrix can be larger than
	// grid -> see explanation above splitCalcDist) is necessary as well as the cuda built in variables
	// which return the block and grid indices to calculate the indices of the vector miming the matrix.
	// See: http://users.wfu.edu/choss/CUDA/docs/Lecture%205.pdf, Slide 16, "Flatten Matrices into linear Memory")
	int Idx_col = threadIdx.x + blockIdx.x*blockDim.x + start[0];
	int Idx_row = threadIdx.y + blockIdx.y*blockDim.y + start[1];
	// Calculating distances between the two atoms acessing the global memory to get thei coordinates.
	//float r = sqrt(pow(x[Idx_col] - x[Idx_row], 2) + pow(y[Idx_col] - y[Idx_row], 2) + pow(z[Idx_col] - z[Idx_row], 2));
	// The following shit is much faster
	float dx = x[Idx_col] - x[Idx_row];
	float dy = y[Idx_col] - y[Idx_row];
	float dz = z[Idx_col] - z[Idx_row];
	float r = dx*dx + dy*dy + dz*dz;
	// Calculate from the interatomic distance the channel/slot of nr which corresponds to the rounded distances value
	// with user defined accuracy/stepsize -> use r again to prevent a new variable definition.
	r = round(sqrt(r)*rp_rstep[0]);
	// Use atomic add inspired by fast histogram: https://devblogs.nvidia.com/parallelforall/gpu-pro-tip-fast-histograms-using-shared-atomics-maxwell/
	// At storage position of array nr corresponding to the calculated distance one is added -> for counting the frequency distances.
	// In case of simoultaneously the acces to one storeage place one theread has to wait until the other has finished.
	// Therefore the bottleneck here is the bandwith and the bus interface for global memory access, but in our case the time determining step
	// is the calculation, so this problem doesen't matter for the 980 GTX TI only an interface load of 5-10 percent is measured and the transfer
	// rate of 346 Gb/s guarantees a very short waiting time in the rare case of simultaniouse access and reduces the probaility of this case.
	// So the speedloss due to this is expected to be insignificant.
	atomicAdd(&nr[int(r)], 1);
}
// Kernel which do tha same as dist2, but checks for phantom atoms before calculation. In case of non defined atom index kernel returns without further
// calculation and is faster available for a new task.
__global__ void care_dist2(int *start, int *end, float *rp_rstep, float *x, float *y, float *z, unsigned int *nr) {
	int Idx_col = threadIdx.x + blockIdx.x*blockDim.x + start[0];
	int Idx_row = threadIdx.y + blockIdx.y*blockDim.y + start[1];
	if (Idx_col > end[0] || Idx_row > end[1]) {
		return;
	}
	float dx = x[Idx_col] - x[Idx_row];
	float dy = y[Idx_col] - y[Idx_row];
	float dz = z[Idx_col] - z[Idx_row];
	float r = dx*dx + dy*dy + dz*dz;
	r = round(sqrt(r)*rp_rstep[0]);
	atomicAdd(&nr[int(r)], 1);
}
// Kernel which do tha same as care_dist2 but for two equal atom subsets, so before the calculation it is checked that only the upper triangular
// distance matrix is calculated (col of subset larger than row of subset).
__global__ void dist(int *start, int *end, float *rp_rstep, float *x, float *y, float *z, unsigned int *nr) {
	int Idx_col = threadIdx.x + blockIdx.x*blockDim.x + start[0];
	int Idx_row = threadIdx.y + blockIdx.y*blockDim.y + start[1];
	if (Idx_col > Idx_row || Idx_row > end[1]) {
		return;
	}
	float dx = x[Idx_col] - x[Idx_row];
	float dy = y[Idx_col] - y[Idx_row];
	float dz = z[Idx_col] - z[Idx_row];
	float r = dx*dx + dy*dy + dz*dz;
	r = round(sqrt(r)*rp_rstep[0]);
	atomicAdd(&nr[int(r)], 1);
}

// Due to watchdog and to display the calculation process it is usefull to split the Grid into loops
// based on given grid and block size, to prevent to long kernel execution times. At a reasonable
// size speed loss do to loops is negligible
// The LoopX and LoopY mimes a grid for the grid, where each part isn't executed parallel but sucessively.
void splitCalcDist(int BlockDimXY, int GridDimXY, int *start_group, int *end_group, float *dev_rp_rstep, float *dev_x, float *dev_y, float *dev_z, unsigned int *dev_nr) {
	// For simplicity a quadratic block size is always used and a quadratic grid size is used to estimate
	// the necessary dimensions of the double loop and only changed for the last column and row loop.
	dim3 block(BlockDimXY, BlockDimXY);
	int GridDimX = GridDimXY;
	int GridDimY = GridDimXY;
	// Calculate number of atoms for subgroup a (columns) and b (rows)  
	int n_col = end_group[0] - start_group[0] + 1;
	int n_row = end_group[1] - start_group[1] + 1;

	// Calculate the loop dimension to cover all atoms of the column subgroup with quadratic grids.
	int LoopDimX = n_col / (GridDimXY*BlockDimXY); // integer division value will be floored
												   // If the number of atoms is a non multiple of GridDimXY*BlockDimXY not all distances will be covered. 
												   // Check this by calculating the remainder after devision
	int LastLoopGridDimX = n_col % (GridDimXY*BlockDimXY);
	if (LastLoopGridDimX != 0) {
		// If not every distance can be covered by a multiple of the defined grid size in X (col) direction,
		// another grid will be added to the X loop dimension and the integer LastLoopGridDimX will be changed
		// from 0 to the number of grids including the last incomplete grid (for a later if condition, so
		// the program knows when the quadratic grid size has to be modified to an rectangular size).
		// Additionally the grid dimension for the last column grid is calculated.
		LoopDimX++;
		LastLoopGridDimX = LastLoopGridDimX / BlockDimXY + 1;
	}
	// Do the same for row subgroup
	int LoopDimY = n_row / (GridDimXY*BlockDimXY);
	int LastLoopGridDimY = n_row % (GridDimXY*BlockDimXY);
	if (LastLoopGridDimY != 0) {
		LoopDimY++;
		LastLoopGridDimY = LastLoopGridDimY / BlockDimXY + 1; // integer devision floors the value so add 1 to ceil and have a grid covering all atoms
	}
	// Display the number of necessary loops to calculate all grids as well as the dimension of the last grid.
	cout << "\nNumber of Row Wise Loops: " << LoopDimY << ", Y-Grid-Dimension for the Last Loop: " << LastLoopGridDimY << endl;
	cout << "Number of Column Wise Loops: " << LoopDimX << ", X-Grid-Dimension for the Last Loop: " << LastLoopGridDimX << endl;
	cout << endl;

	// Introduce a boolean variable for the double loop so always a efficient kernel without
	// if conditions is called if this isn't necessary
	bool careX, careY;
	// Introduce host and device variables to storing the indices of atoms for each grid
	// -> this is necessary because the atom subgroups has to be splitted in subsets fullfilling
	// the conditions of the grid and block sizes
	int *start_subset = new int[2];
	int *end_subset = new int[2];
	int *dev_start_subset, *dev_end_subset;
	gpuErrchk(hipMalloc((void **)&dev_start_subset, 2 * sizeof(int)));
	gpuErrchk(hipMalloc((void **)&dev_end_subset, 2 * sizeof(int)));
	// Start loop over grids rowwise
	for (int LoopY = 0; LoopY < LoopDimY; LoopY++) {
		// Check if the current loop is the last rowwise loop for grid/subset calculation and set careY
		// to 1 and change the row grid dimension if it is the last loop.
		if (LoopY == LoopDimY - 1 && LastLoopGridDimY != 0) {
			GridDimY = LastLoopGridDimY;
			careY = 1;
		}
		else {
			GridDimY = GridDimXY;
			careY = 0;
		}
		// Start loop over columns for actual row
		for (int LoopX = 0; LoopX < LoopDimX; LoopX++) {
			// Check if the current loop is the last columnwise loop for grid/subset calculation and set careY
			// to 1 and change the column grid dimension if it is the last loop.
			if (LoopX == LoopDimX - 1 && LastLoopGridDimX != 0) {
				GridDimX = LastLoopGridDimX;
				careX = 1;
			}
			else {
				GridDimX = GridDimXY;
				careX = 0;
			}
			// Define the grid dimension for the actual row and column the double loop.
			dim3 grid(GridDimX, GridDimY);
			// Calculate the start and end indicies for the current column atom subset wich are parts of the current subgroups.
			start_subset[0] = start_group[0] + LoopX*GridDimXY*BlockDimXY;
			end_subset[0] = start_subset[0] + GridDimX*BlockDimXY - 1;
			// Check if calculated end of subset isn't larger than the end of subgroup -> necessary for last loop 
			// because grid can be a little to large due to ceil (see calculation of LastLoopGridDim).
			if (end_subset[0]>end_group[0]) { end_subset[0] = end_group[0]; }
			// Do the same  for the current row subset
			start_subset[1] = start_group[1] + LoopY*GridDimXY*BlockDimXY;
			end_subset[1] = start_subset[1] + GridDimY*BlockDimXY - 1;
			if (end_subset[1]>end_group[1]) { end_subset[1] = end_group[1]; }
			// Display current loop/grid (including size) and the Atom indices for which the distances are calculated
			cout << "Loop: " << LoopY << "," << LoopX << " -> Grid: " << grid.y << "x" << grid.x << " -> Atom Index: ";
			cout << start_subset[1] << "-" << end_subset[1] << ", ";
			cout << start_subset[0] << "-" << end_subset[0] << endl;
			// copy the atom indices of the current subset from the host to the device
			gpuErrchk(hipMemcpy(dev_start_subset, start_subset, 2 * sizeof(int), hipMemcpyHostToDevice));
			gpuErrchk(hipMemcpy(dev_end_subset, end_subset, 2 * sizeof(int), hipMemcpyHostToDevice));
			// Check some conditions to choose an efficient kernel function for distance calculation
			if (start_group[0] == start_group[1]) {
				// If the distances are calculated between the same subset (column atoms = row atoms) a kernel with permanent
				// if condition before calculation is launched to check that only the upper triangular distance matrix is
				// calculated. The if conditions which are executed for every kernel cost a lot of time but reduce the amount
				// of calculation to a half, which saves more time than necessary for the conditioning.
				dist <<<grid, block>>>(dev_start_subset, dev_end_subset, dev_rp_rstep, dev_x, dev_y, dev_z, dev_nr);
			}
			else if (careX == 0 && careY == 0) {
				// Standard calculation kernel for different subsets, where the row or column of the grid is quadratic as
				// defined by the user and no phantom atoms exist. This is usually the case if the last row or column loop/grid
				// isn't reached or even in case of the last loop if the subgroup was a multiple of block and grid size.
				// Becaus this is checked once before kernel launch, this kernel can run without multiple called if condition 
				// before distance calculation, which saves a lot of computational time since this operation has to be called for 80
				// 90 percent of disctances which are calculated. In case of several billion distances this can save a lot of time.
				// For smaller clusters the effect will be negligible.
				dist2 <<<grid, block>>>(dev_start_subset, dev_rp_rstep, dev_x, dev_y, dev_z, dev_nr);
			}
			else {
				// In case that the grid is larger than the actual atoms in row or column (is only used if a last loop
				// with non user defined grid size is necessary) an if conditioned kernel is laucnched which will test
				// before calculation if the atom truly exists or if it is an empty place (phantom atom) caused by the discrete increments
				// of the block size.
				care_dist2 <<<grid, block>>>(dev_start_subset, dev_end_subset, dev_rp_rstep, dev_x, dev_y, dev_z, dev_nr);
			}
			// Synchronize device to force the host to wait the kernel is finis�hed before a new is launched, otherwise we would get
			// an communication overhead.
			gpuErrchk(hipDeviceSynchronize());
		}
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// FUNCTION TO REDUCE HISTOGRAM OF SUBGROUP SET AND CHECK FOR ERRORS IN CALCULATION
////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// Removes every entry of r and nr where nr=0 (squeeze histogram) and converts squeezed r and nr to double for 
// further calculations which require double precision (Debye double sum). Therfore thrust operations are used
// these are not as efficient as raw cuda but are easier to use and for the moment fast enough.
// ....
// Further informations according thrust see comments in thrust.cu
void reduceCheck(double *dev_r, unsigned int *dev_nr, int channels, int *start_group, int *end_group, unsigned int *dev_nrTmp, double *dev_r_reduced, double *dev_nr_reduced, int *channels_reduced, int row, int col, char *filename, int hist) {
	// Wrap begin of device arrays with thrust vector pointer trD_ indidcates the first storage place of the first value of the thrust vector,
	// therefore the type of stored values is necessary, so thrust knows how many bytes have to be read for one vector element.
	// tdR_+1 will point to the beginning byte of the second value.
	// Because these are only wraps modifing trD via thrust operations actually modifies the device arrays -> so basically trD_nr is the same as dev_nr.
	thrust::device_ptr<double> trD_r(dev_r);
	thrust::device_ptr<unsigned int> trD_nr(dev_nr);
	thrust::device_ptr<double> trD_r_reduced(dev_r_reduced);
	thrust::device_ptr<unsigned int> trD_nrTmp(dev_nrTmp);
	// Get maximum and minimum frequencies as well es their positions stored in a new thrust device vector.
	// Since the wrapped arrays contains no size information start (trD_nr) and endpoint (trD+number_of_elements) are necessary for vector operations.
	thrust::device_ptr<unsigned int> minimum = thrust::min_element(trD_nr, trD_nr + channels); // vector contains all minimum values (multiple times for each occurance)
	thrust::device_ptr<unsigned int> maximum = thrust::max_element(trD_nr, trD_nr + channels);
	// Get the value of minimum and maximum since they are all the same, just use first value.
	unsigned int min_value = minimum[0];
	unsigned int max_value = maximum[0];
	// Calculation of the position of the first occurance of minimum and maximum.
	int min_pos = minimum - trD_nr;
	int max_pos = maximum - trD_nr;
	// First check for errors with output of maximum and minimum values for user interpretation
	if (min_value < 0 || max_value < 0 || min_pos < 0 || min_pos >= channels || max_pos < 0 || max_pos >= channels) {
		cout << "WARNING: Something seems wrong with the following min or max values" << endl;
	}
	cout << "\nMinimum Frequency of Distances: " << min_value << " at " << min_pos << endl;
	cout << "Maximum Frequency of Distances: " << max_value << " at " << max_pos << endl;

	// Find indices of nonzero elements and copy the values to an reduced array
	// ...
	// Create vector to store index values for nonzero elements, which can be dynamically resized in contrary to a pointer vector.
	// But here we will use just a vector of the size of channels and do no resizing.
	// In contrary to a device pointer vector .begin() and .last() can be used to get storage place of first and last value. 
	// So basically the vector know its own length, it is smarter than a pointer vector, but passing it to function you have to pass the complete vector.
	thrust::device_vector<int> trD_Idx(channels);
	// Counting iterator from 1 (first) to channels (last) -> no vector but can be used as vector (creates values on the fly -> saves memory)
	thrust::counting_iterator<int> first(0);
	thrust::counting_iterator<int> last = first + channels;
	// Create an iterator called index iterator, this iterator can be used to count the copied values of an operation to another vector.
	typedef thrust::device_vector<int>::iterator IndexIterator;
	// Copy values from pseudo vector (first to last) in case that corresponding values of vector trD_nr are nonzero to trD_Idx
	// and store the storage place of the last copied number to vector trd_Idx as trD_Idx_end.
	IndexIterator trD_Idx_end = thrust::copy_if(first, last, // vector which is copied indicated by start and end
		trD_nr, // vector on which if condition is applied (should be the same length as the vector from which the values are copied)
		trD_Idx.begin(), // vector where the values are copied
		thrust::identity<unsigned int>()); // condition
										   // Calculate number of copied/nonzero elements which corresponds to unique distances and display them.
	channels_reduced[0] = trD_Idx_end - trD_Idx.begin();
	cout << "Number of Unique Distances: " << channels_reduced[0] << endl;
	// Copy values of r and nr for all indices of nonzero elements stored in trD_Idx to reduced device arrays. 
	// Actually they have the same size as r and nr reduced and every values beyond channels_reduced[0] is treated as data waste.  
	thrust::gather(trD_Idx.begin(), trD_Idx_end, trD_r, trD_r_reduced);
	thrust::gather(trD_Idx.begin(), trD_Idx_end, trD_nr, trD_nrTmp);
	// Copy the values of the reduced arrays containing the distance and the frequencies to the host.
	// Sum all frequencies to get the number of all distances from the subgroup set and convert nr_reduced from int to double
	// for further calculations (Debye double sum).
	double *r = new double[channels_reduced[0]];
	unsigned int *nr = new unsigned int[channels_reduced[0]];
	double *nr_reduced = new double[channels_reduced[0]];
	gpuErrchk(hipMemcpy(r, dev_r_reduced, channels_reduced[0] * sizeof(double), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(nr, dev_nrTmp, channels_reduced[0] * sizeof(int), hipMemcpyDeviceToHost));
	long long unsigned int nr_sum = 0; // for larger cluster number of all distances cause a integer overflow, therefore use int64
	for (int i = 0; i < channels_reduced[0]; i++) {
		nr_sum = nr_sum + long long unsigned int(nr[i]);
		nr_reduced[i] = double(nr[i]);
	}

	// Calculate and display number of unique values -> only for development process
	//unsigned int *nr_all = new unsigned int[channels];
	//gpuErrchk(hipMemcpy(nr_all, dev_nr, channels * sizeof(int), hipMemcpyDeviceToHost));
	//long long unsigned int nr_sum_all = 0;
	//int uniqueValues = 0;
	//for (int i = 0; i < channels; i++){
	//	if (nr_all[i] != 0){
	//		nr_sum_all = nr_sum_all + long long unsigned int(nr_all[i]);
	//		uniqueValues++;
	//	}
	//}

	// Simple error check for distance calculation.
	// ...
	// Calculate number of distances from subgroup set, and compare with nr_sum if thy don't match an integer overflow occured -> Cluster too large
	// for calculation or some other error. Same for the number of atoms and the observed zero distances.
	gpuErrchk(hipMemcpy(dev_nr_reduced, nr_reduced, channels_reduced[0] * sizeof(double), hipMemcpyHostToDevice));
	long long unsigned int n_atoms_col = long long unsigned int(end_group[0] - start_group[0] + 1);
	long long unsigned int n_atoms_row = long long unsigned int(end_group[1] - start_group[1] + 1);
	cout << "Number of Atoms in Row: " << n_atoms_row << endl;
	cout << "Number of Atoms in Column: " << n_atoms_col << endl;
	long long unsigned int nDistTh;
	int j, zeroDist;
	if (start_group[0] == start_group[1]) {
		nDistTh = (n_atoms_col*n_atoms_row - n_atoms_row) / 2 + n_atoms_col;
		if (nDistTh != nr_sum) {
			cout << "WARNING: " << nDistTh << " Distances Expected and " << nr_sum << " Observed!!!" << endl;
		}
		j = 0;
		zeroDist = 0;
		while (r[j] <= 0.0005) {
			zeroDist = zeroDist + nr[j];
			j++;
		}
		if (r[0] != 0 || nr[0] != n_atoms_col) {
			cout << "WARNING: " << "Number of Expected Zero Distances Missing or not Correct!!!" << endl;
			cout << "Number of Atoms within a Tolerance of 0.0005 A is " << zeroDist << endl;
		}
		while (r[j] < 0.25) {
			zeroDist = zeroDist + nr[j];
			j++;
		}
		if (zeroDist > n_atoms_col) {
			cout << "WARNING: " << "Possible Atom Collision!!! Found around " << zeroDist - n_atoms_col << " interatomic Distances lower than the Radius of an Hydrogen Atom." << endl;
		}
		cout << "Atoms Expected: " << n_atoms_col << " - Atoms Observed: " << nr[0] << endl;
	}
	else {
		nDistTh = n_atoms_col * n_atoms_row;
		if (nDistTh != nr_sum) {
			cout << "WARNING: " << nDistTh << " Distances Expected and " << nr_sum << " Observed!!!" << endl;
		}
		j = 0;
		zeroDist = 0;
		while (r[j] < 0.25) {
			zeroDist = zeroDist + nr[j];
			j++;
		}
		if (zeroDist > 0) {
			cout << "WARNING: " << "Possible Atom Collision!!! Found around " << zeroDist << " interatomic Distances lower than the Radius of an Hydrogen Atom." << endl;
		}
	}
	cout << "Distances Expected: " << nDistTh << " - Distances Observed: " << nr_sum << endl;


	// Writing the calculated diffraction data to an output file
	if (hist > 0) {
		clock_t time_start = clock();
		stringstream savefile; // convert the name of the input file to a stringstream
		savefile << filename << "x" << row << "x" << col; // append an I for 'Intensity' to this name to create the name of the output file
		cout << "\nWriting Distance File " << savefile.str() << endl;
		ofstream myfile; // Create an outputstrem named myfile
		myfile.open(savefile.str()); // use open property of outputstream to create/open the *.debI file -> this name is achieved by using the string property of the stringstream
		for (int i = 0; i < channels_reduced[0]; i++) { // write output to myfile instead to the cout (CommandWindow Out)
			myfile << setprecision(7) << r[i] << "\t" << nr[i] << endl;
		}
		myfile.close(); // close the file
						// Get end time for file reading
		clock_t time_end = clock();
		// Calculate and display time for reading the input file
		float elapsed_time = float(time_end - time_start) / CLOCKS_PER_SEC;
		cout << "Time for Writing Distance File: " << elapsed_time << " s" << endl;
	}


	// Synchronize device to force the host to wait the kernel is finis�hed before a new is launched, otherwise we would get
	// an communication overhead.
	gpuErrchk(hipDeviceSynchronize());
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// KERNEL AND CONTROLLING FUNCTION TO CALCULATE SCATTERING PREFACTOR OF A SUBGROUP SET
////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// I = ffoobb * sum(nr*sin(Kr)/Kr) .... simple sum over distances is the debye double sum over all atoms.
// Kerrnel function which gets atomic scattering factor, occupancy and isotropic thermal displacement (debye-waller) to calculate the factor
// ffoobb = fi*fj*Oi*Oj*exp[-Mi]*exp[-Mj] for every K/TwoTheta.
// M = 8*pi�*u�*sin�(theta)/lambda� = B*sin�(theta)/lambda�, B = 8*pi�
__global__ void atomicScatter(int type1, int type2, int size_K, double *occ, double *beq, double *K, double *a, double *b, double *c, double *ffoobb) {
	// Kernel is executed for each K/TwoTheta (one dimensional grid)
	int Idx = blockIdx.x*blockDim.x + threadIdx.x;
	// Only execute if K/TwoTheta exists and is no phantom value, caused be discrete grid and block size.
	if (Idx < size_K) {
		double rp16pi2 = -0.006332573977646; // = (-1) * 1/(16*pi�)
		double negativeHalfSquaredS = K[Idx] * K[Idx] * rp16pi2; // = -sin�(theta)/lambda�, s = 2*sin(theta)/lambda = 1/d
																 // Calculate occupancy and debye-waller part of the prefactor
		ffoobb[Idx] = occ[type1] * occ[type2];
		ffoobb[Idx] = ffoobb[Idx] * exp(negativeHalfSquaredS*(beq[type1] + beq[type2]));
		// Calculate atomic scattering factords from 11 parameter approximation.
		double f1 = c[type1];
		double f2 = c[type2];
		for (int i = 0; i < 5; i++) {
			f1 += a[type1 * 5 + i] * exp(b[type1 * 5 + i] * negativeHalfSquaredS);
			f2 += a[type2 * 5 + i] * exp(b[type2 * 5 + i] * negativeHalfSquaredS);
		}
		// Complement prefactor with calculated scattering factors
		ffoobb[Idx] = ffoobb[Idx] * f1*f2;
	}
}
// Controlling function to determine 1D block and grid size based one user defined 2D grid and block sizes
void atomicProp(int BlockDimXY, int type1, int type2, int nTT, double *dev_occ, double *dev_beq, double *dev_K, double *dev_a, double *dev_b, double *dev_c, double *dev_ffoobb) {
	dim3 block(BlockDimXY*BlockDimXY);
	dim3 grid(nTT / block.x + 1);
	cout << "\nCalculation of Atomic Properties" << endl;
	cout << "Grid: " << grid.y << "x" << grid.x << "\t";
	cout << "Block: " << block.y << "x" << block.x << endl;
	atomicScatter <<<grid, block>>>(type1, type2, nTT, dev_occ, dev_beq, dev_K, dev_a, dev_b, dev_c, dev_ffoobb);
	// Synchronize device to force the host to wait the kernel is finis�hed before a new is launched, otherwise we would get
	// an communication overhead.
	gpuErrchk(hipDeviceSynchronize());
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// KERNEL AND CONROLLING FUNCTION FOR A ROWSUM OF AN IMAGINARY MATRIX
////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// Kernel to calculate the row sum of the following matrix to get the intensitys for each K/TwoTheta
//         r[0]    r[1]    r[2]  ....      =     Row Sum
// K[0]    I[0,0]  I[0,1]  I[0,2]				 Isum[0]
// K[1]    I[1,0]  I[1,1]  I[1,2]                Isum[1]
// K[3]    I[2,0]  I[2,1]  I[2,2]                Isum[2]
//   .
//   .
// ...
__global__ void debyeRowSum(int Kstart, int Kend, int size_nr, double *K, double *r, double *nr, double *ffoobb, double *Isum) {
	// Create an array to store double values in size of the block size (=BlockDimXY�) this value has to be set an given by the 
	// controlling function, if the size is define here as __shared__double partialSum[64] only a specific number and no variable can be used.
	extern __shared__ double partialSum[];
	// Calculate the one dimensional thread index as the block would be read column by column, because the shared array is only 1D
	int tid = threadIdx.y*blockDim.x + threadIdx.x;
	// Calculate row and column of the current thread, because the block is actually two dimensional. For the column Kstart isn't necessary, see
	// controlling function (grid is 1D as well as loop)
	int Idx_col = blockIdx.x*blockDim.x + threadIdx.x; // only for partial sum, these values will be added later
	int Idx_row = Kstart + blockIdx.y*blockDim.y + threadIdx.y; // row indicates which K/TwoTheta value is used
																// Check if K/TwoTheta exist or phantom value was generated by the discrete block and grid size.
	if (Idx_row <= Kend) {
		// Check if the first BlockDimX (col) values of interatomic distances are above 0.0005 A than it will be treated as distance between two different
		// atoms and if the value is below the program will assume it is the same atom and the deviation from zero was caused by a rounding error.
		// According to this calculate the value of the debye equation for the current K and r set and store it in the shared memory for the  current block.
		if (r[Idx_col] > 0.0005) {
			partialSum[tid] = 2 * nr[Idx_col] * sin(K[Idx_row] * r[Idx_col]) / (K[Idx_row] * r[Idx_col]); // times two because only distances of upper triangular distance matrix where calculated
		}
		else
		{
			partialSum[tid] = nr[Idx_col]; // sin(0)/0 = 1 so nr*sin(0)/0 = nr
		}
		// Since the grid dimension for columns is 1 only the values for r/nr up to BlockDimX would be calculated, therefore in this thread 
		// a for loop is started which adds on this inital value the block dimension until all distances are covered, which would have the
		// same threadIdx for further imaginary blocks in X (col) direction if the grid would be two dimensional. 
		// These values are just added to the partial sum of imaginary 1D thread tid. (see additional comments and schemes, RowSumSheme)
		Idx_col += blockDim.x;
		for (Idx_col; Idx_col < size_nr; Idx_col += blockDim.x) {
			partialSum[tid] += 2 * nr[Idx_col] * sin(K[Idx_row] * r[Idx_col]) / (K[Idx_row] * r[Idx_col]);
		}
	}
	// Wait until all threads have calculated the partial sum 
	__syncthreads();
	// For a sumation of rows only one thread is necessary for each row, so choose only threads with threadIdx.x=0 (first column in block).
	// Then simply sum the contents within a row of the block -> the result will be the complete row sum. 
	if (threadIdx.x == 0 && Idx_row <= Kend) {
		double Sum = partialSum[tid];
		for (int i = 1; i < blockDim.x; i++) {
			Sum += partialSum[tid + i];
		}
		// Calculate Ipart for current subgroup set
		Sum = Sum*ffoobb[Idx_row];
		// Add this value to the complet summed intensity for all atoms.
		Isum[Idx_row] += Sum;
	}
}

// Controlling function for row sum kernel to set block, grid and shared memory size and splits the calculastion/grid into 
// some loops to avoid to long kernel execution times (same procedure as in splitCalcDist but only one dimensional).
void intensity(int BlockDimXY, int GridDimXY, int size_K, int size_nr, double *dev_K, double *dev_r_reduced, double *dev_nr_reduced, double *dev_ffoobb, double *dev_Isum) {
	// Convert size integers in uint64 to prevent integer overflow during the calculation of LoopDimY.
	long long unsigned int uint64_size_K = long long unsigned int(size_K);
	long long unsigned int uint64_size_nr = long long unsigned int(size_nr);
	long long unsigned int uint64_BlockDimXY = long long unsigned int(BlockDimXY);
	long long unsigned int uint64_GridDimXY = long long unsigned int(GridDimXY);
	// Set the size of shared memory so that each thread of a block can store one double value in shared memory
	size_t sharedMem = BlockDimXY*BlockDimXY * sizeof(double);
	// Calculate the loop and grid dimension for rows, no dimension is set in X (col) direction, so that one block can do the complete row summation.
	// For this the user defined input size for an quadratic grid and block size is used.
	int LoopDimY = int((uint64_size_nr*uint64_size_K) / (uint64_BlockDimXY*uint64_BlockDimXY*uint64_GridDimXY*uint64_GridDimXY)) + 1;
	int GridDimY = (size_K / BlockDimXY + 1) / LoopDimY + 1;
	dim3 block(BlockDimXY, BlockDimXY); // block dimension is 2D to take advantage of shared memory for faster operation
	dim3 grid(1, GridDimY);
	// Show calculated loop and grid sizes
	cout << "\nEvaluation of the Debye Double Sum" << endl;
	cout << "Number of Unique Distances: " << size_nr << endl;
	cout << "LoopDimY: " << LoopDimY << " -> Grid: " << grid.y << "x" << grid.x << endl;
	// Introduce variables which store the start and end index for current loop/grid of K/TwoTheta 
	int start = 0;
	int end = 0;
	for (int LoopY = 0; LoopY < LoopDimY; LoopY++) {
		// Calculate start and end index of current grid and check if end index is larger than number of K/TwoTheta.
		start = LoopY*GridDimY*BlockDimXY;
		end = start + GridDimY*BlockDimXY - 1;
		if (end > size_K - 1) { end = size_K - 1; }
		// Display current loop and which rows/K are summed.
		cout << "Loop: " << LoopY << "\tK-Index: " << start << "-" << end << endl;
		// Launch kernel to calculate the matrix K[start:end] x r[all] and sum the rows. Beside grid and block size, the size of shared memory has
		// to be passed to the kernel as third value (for this a size_t type is necessary).
		debyeRowSum <<<grid, block, sharedMem>>>(start, end, size_nr, dev_K, dev_r_reduced, dev_nr_reduced, dev_ffoobb, dev_Isum);
		// Synchronize device to force the host to wait the kernel is finis�hed before a new is launched, otherwise we would get
		// an communication overhead.
		gpuErrchk(hipDeviceSynchronize());
	}
}



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// BEGIN OF MAIN FUNCTION (READ FILE; STORE ATOM POSITIONS; ALLOCATE AND MEMORY ON GPU; COPY THE IMPORTANT DATA TO THE DEVICE; INITIATE CALCULATION)
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// Mostly pointers (indicated by *) are used to store values to avoid passing large arrays between functions
// and avoid problems with the device storage.
// Excample:
// int *E = new int[x] creates an 1D array with x storage places for integers (size: 4*x bytes)
// - The firstt four storage adresses contain the first value (the adresses of on pointer array are side by side)
// - Calling E will return the storage adress of the first value
// - Calling E[0] will return the first value
// - Calling &E[x-1] will return the first adress of the four storage adresses (4 bytes) of the last value in the pointer
//   -> the and will give the adresses even for normal declared variables
// - Overadressing E[x]=a or writing a on storage adress E+4*x will cause a manipulation of a other variable
//   or array depending on system reservation -> this could be a variable from a comlete other program or the 
//   System, which can cause a crash or of an own program variable, which will totally mess up any further 
//   calculation.

void main(int argc, char* argv[])
//void main()
{
	// Display Information
	cout << "\ncuDebye - Version: 1.5" << endl;
	cout << " - Calculation of distances with square root in the kernel function" << endl;
	cout << " - Writes intensity and distance files" << endl;
	cout << "-----------------------------------------------------------------------------" << endl;
	cout << "Author: Martin Rudolph" << endl;
	cout << "Technische Universit\x84t Bergakademie Freiberg" << endl;
	cout << "Institute of Materials Science" << endl;
	cout << "Gustav-Zeuner-Strasse 5" << endl;
	cout << "09599, Freiberg, Germany" << endl;
	cout << "E-Mail: m.s.rudolph@outlook.com\n" << endl;
	// Get FileName from the argument vector
	char *filename = argv[1];
	//char filename[] = "C:\\Users/rudolp2/Desktop/GammaAl2O3_10nm.deb";
	// Start timer for measuring time to load file
	clock_t time_start = clock();
	// Call function to check if input file exists (this function returns a boolean)
	if (fexists(filename) == 0) {
		cout << "\nERROR: >>" << filename << "<< does not exist" << endl;
		return;
	}
	// Define known variables stored in the first three lines of the input file
	double lambda, TwoThetaMin, TwoThetaMax, TwoThetaStep;
	double rmax, rstep;
	float rp_rstep;
	int CudaDevice, BlockDimXY, GridDimXY, hist, subgroups;
	// Create the reading stream "file" and initialize a string for a line by line scan
	ifstream file(filename);
	string line;

	// Before and in the header there should be no empty lines.

	// Get String of the first line and convert this string to an own temporary stream
	getline(file, line);
	istringstream tmp(line);
	// Extract the Index of the CUDA device which will be used for the calculation as well as the user
	// defined block and grid size in order to guard against the overstrain of an older GPU or to prevent 
	// watchdog crashes if the programm runs on windows and a display is connected to the selected 
	// calculating device or a kernel execution timeout is enabled.
	// If only one nVidia card is installed the device index is typically zero.
	// WARNING: THIS PROGRAMM CAN ONLY USE ONE GPU/DEVICE FOR CALCULATION; THE INDEX DOES NOT 
	// SPECIFY THE NUMBER OF PARALLEL COMPUTING GPUs; IT DEFINES THE EXECUTING GPU !!!
	tmp >> CudaDevice >> BlockDimXY >> GridDimXY;
	// Define the calculating cuda device specified by the input file
	gpuErrchk(hipSetDevice(CudaDevice));
	// Initiate a structure to store the properties of an cuda device
	hipDeviceProp_t prop;
	// Get the properties of the choosen calculating device
	gpuErrchk(hipGetDeviceProperties(&prop, CudaDevice));
	// Because the free memory of the devices isn't stored in the properties structure initiate two additional
	// variables to get the free device memory.
	size_t mem_free = 0;
	size_t mem_tot = 0;
	gpuErrchk(hipMemGetInfo(&mem_free, &mem_tot));
	// Display the important device properties
	cout << "\nCUDA PROPERTIES" << endl;
	cout << "----------------------------------------------------------------------------\n" << endl;
	cout << "Index of Selected Calculating Device: " << CudaDevice << endl;
	cout << "Number of Streaming Multiprocessors: " << prop.multiProcessorCount << endl;
	cout << "Clock Rate: " << prop.clockRate << " KHz" << endl;
	cout << "Kernel Timeout Enabled: " << prop.kernelExecTimeoutEnabled << endl;
	// Thread and block properties of the device
	cout << "\nMax Threads Per Block: " << prop.maxThreadsPerBlock << endl;
	cout << "Max Threads per Block in X-Dimension: " << prop.maxThreadsDim[0] << endl;
	cout << "Max Threads per Block in Y-Dimension: " << prop.maxThreadsDim[1] << endl;
	cout << "Threads per Warp: " << prop.warpSize << endl; // a block is actually splited into warps, therfore the size of an block should be ideally a multiple of 32
	cout << "\nMax Blocks per Grid in X-Dimension: " << prop.maxGridSize[0] << endl;
	cout << "Max Blocks per Grid in Y-Dimension: " << prop.maxGridSize[1] << endl;
	cout << "\nNote: in Cuda the X describes the Column and the Y the Row. ";
	cout << "As it is typical in C, Python and Matlab for displaying the Grid and Block Sizes ";
	cout << "the following format is used: Rows x Columns." << endl;
	// Available memory
	cout << "\nTotal Global Memory: " << prop.totalGlobalMem << " byte" << endl;
	cout << "Free Available Memory: " << mem_free << " byte" << endl;
	cout << "Shared Memory per Block: " << prop.sharedMemPerBlock << " byte" << endl;
	// User defined block and grid size
	cout << "\nUSER DEFINED GRID AND BLOCK SIZE" << endl;
	cout << "----------------------------------------------------------------------------\n" << endl;
	cout << "Block: " << BlockDimXY << "x" << BlockDimXY << "(" << BlockDimXY*BlockDimXY << ")";
	cout << "\tGrid: " << GridDimXY << "x" << GridDimXY << "(" << GridDimXY*GridDimXY << ")" << endl;

	// Get string from the second line, clear the temporary string stream and load the new string into this
	// stream. Extract from this stream the wavelength the lower and upper limit for 2Theta as well as the
	// step to specify for which range the diffractogram  is calculated.
	getline(file, line);
	tmp.clear();
	tmp.str(line);
	tmp >> lambda >> TwoThetaMin >> TwoThetaMax >> TwoThetaStep;
	int nTT = int((TwoThetaMax - TwoThetaMin) / TwoThetaStep) + 1; // number of TwoTheta discrete TwoTheta values
																   // Create pointers to store TwoTheta and K at the Host
	double *TwoTheta = new double[nTT];
	double *K = new double[nTT];
	// Calculate TwoTheta as well as K from the given input values
	for (int i = 0; i < nTT; i++) {
		TwoTheta[i] = TwoThetaMin + double(i)*TwoThetaStep;
		K[i] = 4 * PI*sin(TwoTheta[i] / 2 * PI / 180) / lambda;
	}
	// Display settings for calculation of the diffractogram
	cout << "\nDIFFRACTOGRAM SETTINGS" << endl;
	cout << "----------------------------------------------------------------------------\n" << endl;
	cout << "Wavelength: " << lambda << " A" << endl;
	cout << "2ThetaMin: " << TwoThetaMin << "\370" << "\t2ThetaMax: " << TwoThetaMax << "\370" << "\t2ThetaStep: " << TwoThetaStep << "\370" << endl;
	cout << "Number of 2Theta values: " << nTT << "\tStart: 0 = " << TwoTheta[0] << "\370" << "\tEnd: " << nTT - 1 << " = " << TwoTheta[nTT - 1] << "\370" << endl;
	cout << "k = 4*pi*sin(theta)/lambda: " << "\tStart: " << K[0] << "\tEnd: " << K[nTT - 1] << endl;

	// Read from the third line the maximum possible distance between two atoms within the cluster
	// as well as the accuracy for the calculation of interatomic distances (in Angstrom). 
	// Usually the rmax value should be a bit larger than the real maximum to prevent a crash in case of 
	// rounding errors. I usually add 5% and round this number up to the next integer. 
	// For the accuracy the perfect value seems to be 0.0001. Be Carefull above 0.001 the diffractogram
	// will get worse and below 0.0001 you will see no real effect due to the single precission calculation
	// and the fact, that atomic position in structure files are in most cases limited to 4 digits due to uncertainties.
	// Anyway it should be noted, that due to the single precision operations to speed up the calculation the
	// efficient accuracy is not 0.0001 it should be around 0.0003 as long as the maximum distance between 
	// two atoms is less than 999 A -> At the time I haven't checked this properly.
	// The issue is:
	// - Single precission allows 7 significant nearly exact digits -> 999.1234
	// - Error = (x*dx+y*dy+z*dz)/sqrt(x^2+y^2+z^2)
	// - Assume x=y=z and dx=dy=dz=0.00005 -> Error sqrt(3)*dx = 0.000087
	// - Rounding Error due to 7-8 significant digits neglecteted for the square sum x^2+y^2+z^2
	//   -> therefore a value around 0.0003 is assumed as worst case error
	// - Clusters above 1000 A can cause larger errors because dx_max rises to nearly 0.0005 due to the
	//   precision of single values as well as the rounding error of the square sum becomes more problematic
	// Please look carefully at your diffractogram for larger clusters.
	getline(file, line);
	tmp.clear();
	tmp.str(line);
	// Extract from the third line string/stream rmax and rstep
	tmp >> rmax >> rstep >> hist;
	// Calculate r and the number of cahnnels/bins to store the frequencies of interatomic distances
	int channels = int(rmax / rstep) + 2;
	double *r = new double[channels];
	for (int i = 0; i < channels; i++) {
		r[i] = double(i)*rstep;
	}
	// Define reciprocal rstep for global parallel device functions because a division within such a 
	// function is problematic and time consuming.
	rp_rstep = float(1 / rstep);
	// Display settings for interatomic distance histogram
	cout << "\nDISTANCE CALCULATION SETTINGS" << endl;
	cout << "----------------------------------------------------------------------------\n" << endl;
	cout << "Rmax: " << rmax << " A\tRstep: " << rstep << " A" << endl;
	cout << "Number of Channels: " << channels << "\tStart: 0 = " << r[0] << " A\tEnd: " << channels - 1 << " = " << r[channels - 1] << " A" << endl;
	if (hist == 1) {
		cout << "Output Distance File: True \t Output Intensity File: False" << endl;
	}
	else if (hist == 2) {
		cout << "Output Distance File: True \t Output Intensity File: True" << endl;
	}
	else {
		hist = 0;
		cout << "Output Distance File: False \t Output Intensity File: True" << endl;
	}

	// Extract the number from the fourth line in file which should contain the number of atomic subgroups of 
	// identical type, occupancy and isotropic temperature factor
	getline(file, line);
	tmp.clear();
	tmp.str(line);
	tmp >> subgroups;
	cout << "\nCLUSTER AND ATOMIC INFORMATIONS" << endl;
	cout << "----------------------------------------------------------------------------\n" << endl;
	cout << "Number of Subgroups: " << subgroups << endl;
	// Based on the number of subgroups the next lines are read. Each line should contain the number of the atoms,
	// the atom type, the occupancy and the isotropic temperature factor and the 11 parameter atomic scattering factor
	// approximation in the subgroup. Beside the extraction of
	// these informations. The indices of atoms for beginning and ending of subgroups are calculateted. This is 
	// necessary, because all the x,y,z-positions will be stored in three arrays for all subgroups together.
	int *n_atoms = new int[subgroups]; // array which stores the atom number of each subgroup
	int *start = new int[subgroups]; // array which stores the atom index where a subgroup starts
	int *end = new int[subgroups]; // array which stores the atom index where a subgroup starts
	string *type = new string[subgroups];
	double *occ = new double[subgroups];
	double *beq = new double[subgroups];
	double *a = new double[subgroups * 5]; // arrays to store scattering factors for all subgroups
	double *b = new double[subgroups * 5];
	double *c = new double[subgroups];
	start[0] = 0; // first subgroup starts at 0
				  // Extract information from every subgroup and calculate the start and end indices of each subgroup and
				  // display them for user control.
	for (int i = 0; i < subgroups; i++) {
		getline(file, line);
		tmp.clear();
		tmp.str(line);
		tmp >> n_atoms[i] >> type[i] >> occ[i] >> beq[i] >> a[i * 5 + 0] >> a[i * 5 + 1] >> a[i * 5 + 2] >> a[i * 5 + 3] >> a[i * 5 + 4] >> b[i * 5 + 0] >> b[i * 5 + 1] >> b[i * 5 + 2] >> b[i * 5 + 3] >> b[i * 5 + 4] >> c[i];
		cout << "\nGroup " << i << " - Number of Atoms: " << n_atoms[i] << endl;
		end[i] = start[i] + n_atoms[i] - 1;
		if (i < subgroups - 1) {
			start[i + 1] = start[i] + n_atoms[i];
		}
		cout << "Atom index from " << start[i] << " to " << end[i] << endl;
		cout << "Atom Type: " << type[i] << ", Occupancy: " << occ[i] << ", Beq: " << beq[i] << endl;
		cout << "Scattering Coefficients:" << endl;
		cout << "a[1-5]: ";
		for (int j = 0; j < 5; j++) {
			cout << a[i * 5 + j] << ", ";
		}
		cout << "\nb[1-5]: ";
		for (int j = 0; j < 5; j++) {
			cout << b[i * 5 + j] << ", ";
		}
		cout << "\nc: " << c[i] << endl;
	}
	int n_atoms_sum = end[subgroups - 1] + 1; // calculate number of all atoms

											  // Based on the total number of atoms three 1D arrays using pointers (*) are created to store the atomic 
											  // positions unsing single precision.
	float *x = new float[n_atoms_sum];
	float *y = new float[n_atoms_sum];
	float *z = new float[n_atoms_sum];
	cout << "\nMemory allocated for " << n_atoms_sum << " Atoms." << endl;
	cout << "Reading Atomic Data..." << endl;
	// Read line by line x, y and z position of the atoms and ignore empty lines, which can be added
	// in the input file to show clearly start and en of a subgroup (not necessary).
	// Be carefull empty lines followed by empty lines as well as to much empty lines at the end
	// of the file can cause a program crash -> Why is not clear at the moment.
	int i = 0;
	while (getline(file, line))
	{
		if (line.empty() == 0) // if line is non empty read atomic positions and stor them
		{
			istringstream tmp(line);
			tmp >> x[i] >> y[i] >> z[i];
			i++;
		}
	}
	// Show number of loaded atoms and check if they match the number specified by the subgroups in the header.
	cout << i << " Atoms are loaded!" << endl;
	if (i != n_atoms_sum) {
		cout << "\n\nERROR: Number of extracted Atoms doesn't match the number specified in the preamble!" << endl;
		return;
	}
	// close file
	file.close();
	// Get end time for file reading
	clock_t time_end = clock();
	// Calculate and display time for reading the input file
	float elapsed_time = float(time_end - time_start) / CLOCKS_PER_SEC;
	cout << "\nTime for Loading the Input File: " << elapsed_time << " s" << endl;

	////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Allocate and Intialize important Variables on the GPU
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Start clock for measuring allocation time
	time_start = clock();
	// Get and show free device memory before memory allocation on the GPU
	cout << "\nALLOCATING THE NECESSARY MEMORY ON THE DEVICE" << endl;
	cout << "----------------------------------------------------------------------------\n" << endl;
	gpuErrchk(hipMemGetInfo(&mem_free, &mem_tot));;
	cout << "Free Memory before Allocation: " << mem_free << " byte" << endl;

	// Initialize pointer adresses stored in the host but pointing to the device memory
	float *dev_rp_rstep, *dev_x, *dev_y, *dev_z;
	double *dev_r_reduced, *dev_nr_reduced, *dev_r, *dev_K, *dev_occ, *dev_beq, *dev_a, *dev_b, *dev_c, *dev_ffoobb, *dev_Isum;
	unsigned int *dev_nr, *dev_nrTmp;
	int *channels_reduced = new int[1]; // pointer for single varible to store the number of reduced channels (see function reduceCheck)

										// Reserve storage place in bytes on the device, with no specific storage type (void**) as typical for the host
										// Therefor the pointer from the host is used and to provide enougth bytes to store different types 
										// additional the number of storage places/bytes have to be passed. This can easiely calculateted from
										// by length_of_array*size_of_type
										// ...
										// Reciprocal accuaracy of distances, float type -> see explaination at atom position allocation on device.  
	gpuErrchk(hipMalloc((void**)&dev_rp_rstep, sizeof(float))); // only one float
																 // Array of discrete interatomic diatances and array to stor their frequencies.
	gpuErrchk(hipMalloc((void**)&dev_r, channels * sizeof(double))); // number of discrete r * 8 byte
																	  // nr as integer for histogram/frequency, since only the number of occuring distances is counted
																	  // via atomic add unsigned integer is necessary for maximum performance. This allows a counting up
																	  // to 2^32 unique distances which should be enougth even for larger clusters.
																	  // For temporary operations an additional variable is created.
	gpuErrchk(hipMalloc((void**)&dev_nr, channels * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_nrTmp, channels * sizeof(int)));
	// Creating reduced variables to store only values where nr!=0.
	// Use same length (channels) as nr for worst case that every distance r occurs at least once.
	// They are stored from index 0 to channels_reduced[0]
	// nr contains only the distances from atom i to j and has to be multiplied with 2 to mimic the distance from j to i.
	// For Calculation of Intensity double precision is necessary (see intensity and debyeRowSum), also in case
	// that nr for one distance is larger than 1/2*2^32 this format is necessary to prevent integer overflow
	// when multiplying with 2.
	gpuErrchk(hipMalloc((void**)&dev_r_reduced, channels * sizeof(double)));
	gpuErrchk(hipMalloc((void**)&dev_nr_reduced, channels * sizeof(double)));
	// K = 4*pi*sin(Theta)/lambds and array to store corresponding intensity
	gpuErrchk(hipMalloc((void**)&dev_K, nTT * sizeof(double))); // nTT -> number of TwoTheta bzw. K values
	gpuErrchk(hipMalloc((void**)&dev_Isum, nTT * sizeof(double)));
	// Atom positions float precision for faster calculation of distance in kernel functions.
	gpuErrchk(hipMalloc((void**)&dev_x, n_atoms_sum * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_z, n_atoms_sum * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_y, n_atoms_sum * sizeof(float)));
	// Occupancies and Beq and atomic scattering factors only necessary for intensity calculation, 
	// therefore double precision necessary.
	gpuErrchk(hipMalloc((void**)&dev_occ, subgroups * sizeof(double)));
	gpuErrchk(hipMalloc((void**)&dev_beq, subgroups * sizeof(double)));
	// Atomic scattering factors
	gpuErrchk(hipMalloc((void**)&dev_a, 5 * subgroups * sizeof(double))); // *5 beacause every subgroup has 5 a parameters
	gpuErrchk(hipMalloc((void**)&dev_b, 5 * subgroups * sizeof(double)));
	gpuErrchk(hipMalloc((void**)&dev_c, subgroups * sizeof(double)));
	// Array to store fi*fj*Occi*Occj*DebWalli*DebWallj for every TwoTheta -> see atomicProp and atomicScatter 
	gpuErrchk(hipMalloc((void**)&dev_ffoobb, nTT * sizeof(double)));

	// Copy host values to the device and set counting values to zero
	gpuErrchk(hipMemcpy(dev_rp_rstep, &rp_rstep, sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_r, r, channels * sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_K, K, nTT * sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(dev_nr, 0, channels * sizeof(int)));
	gpuErrchk(hipMemset(dev_Isum, 0, nTT * sizeof(double)));
	gpuErrchk(hipMemcpy(dev_x, x, n_atoms_sum * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_y, y, n_atoms_sum * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_z, z, n_atoms_sum * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_occ, occ, subgroups * sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_beq, beq, subgroups * sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_a, a, 5 * subgroups * sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_b, b, 5 * subgroups * sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_c, c, subgroups * sizeof(double), hipMemcpyHostToDevice));

	// Get free memory of device after allocation and display it.
	gpuErrchk(hipMemGetInfo(&mem_free, &mem_tot));
	cout << "Free Memory after Allocation: " << mem_free << " byte" << endl;
	// Calculate and display time for allocation
	time_end = clock();
	elapsed_time = float(time_end - time_start) / CLOCKS_PER_SEC;
	cout << "\nTime for Allocating the necessary Memory on the Device: " << elapsed_time << " s" << endl;

	////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Calculation of each Subgroup Combination (Distances+Intensity)
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Start timer for actual calculation
	time_start = clock();
	cout << "\n\n----------------------------------------------------------------------------" << endl;
	cout << "START OF ACTUAL CALCULATION" << endl;
	cout << "----------------------------------------------------------------------------" << endl;
	// Define variables to store start and end of atom indices for current subgroup combination. 
	int *current_group_start = new int[2];
	int *current_group_end = new int[2];
	// Run double loop over each possible subroup combination, imitating a matrix via loops
	//     0    1    2    3
	// 0  00   01   02   03
	// 1       11   12   13
	// 2            22   23
	// 3                 33
	for (int row = 0; row < subgroups; row++) {
		for (int col = row; col < subgroups; col++) {
			// Get start and end atom index for subgroup combination of the pseudo matrix 
			current_group_start[0] = start[col]; // col in cuda X
			current_group_start[1] = start[row]; // row in cuda Y
			current_group_end[0] = end[col];
			current_group_end[1] = end[row];
			// Display the current subgroup for Calculation
			cout << "\n\n>>>> Calculation of Subgroup Number " << row << " and " << col << endl;
			// Calculate the histogram of distances -> most time consuming process
			splitCalcDist(BlockDimXY, GridDimXY, current_group_start, current_group_end, dev_rp_rstep, dev_x, dev_y, dev_z, dev_nr);
			// Reduce the arrays r and nr to r_reduced and nr_reduced in combination with an error check for 
			// the calculation. (could be improved but it is fast enough compared to the splitCalc and intensity) 
			reduceCheck(dev_r, dev_nr, channels, current_group_start, current_group_end, dev_nrTmp, dev_r_reduced, dev_nr_reduced, channels_reduced, row, col, filename, hist);
			if (hist != 1) {
				// Calculate the preamble of each subgroup for TwoTheta containign the atomic scattering factors,
				// the occupancies as well as the isotropic temperature factors. (could be improved but it is fast enough compared to the splitCalc and intensity)
				atomicProp(BlockDimXY, row, col, nTT, dev_occ, dev_beq, dev_K, dev_a, dev_b, dev_c, dev_ffoobb);
				// Calculate the portiton of scattered intensity from the actual subgroup combinations, and add
				// these part to the intensity of the previouse parts -> second most time consuming operation,
				// which is small compared to splitCalc but increse as the number of unique distances increases.
				intensity(BlockDimXY, GridDimXY, nTT, channels_reduced[0], dev_K, dev_r_reduced, dev_nr_reduced, dev_ffoobb, dev_Isum);
				// Reset histogram counting array for next subgroup combination.
			}
			gpuErrchk(hipMemset(dev_nr, 0, channels * sizeof(int)));
		}
	}
	// Calculate and display the time for the time intense calculation stuff
	time_end = clock();
	elapsed_time = float(time_end - time_start) / CLOCKS_PER_SEC;
	cout << "\nTime for Calculation of Distances and\nEvaluation of the Debye Double Sum: " << elapsed_time << " s" << endl;
	if (hist != 1) {
		// Create host variable and transfer the calculated intensity from the device to the host.
		double *Isum = new double[nTT];
		gpuErrchk(hipMemcpy(Isum, dev_Isum, nTT * sizeof(double), hipMemcpyDeviceToHost));
		// Display the first 10 and last 10 values of the calculated diffraction data.
		cout << "\n\n" << endl;
		cout << "2Theta\tIntensity" << endl;
		cout << "------------------" << endl;
		for (int i = 0; i < 10; i++) {
			cout << TwoTheta[i] << "\t" << Isum[i] << endl;
		}
		cout << "\n...\n" << endl;
		for (int i = nTT - 11; i < nTT; i++) {
			cout << TwoTheta[i] << "\t" << Isum[i] << endl;
		}

		// Writing the calculated diffraction data to an output file
		time_start = clock();
		cout << "\n Writing Output File..." << endl;
		stringstream savefile; // convert the name of the input file to a stringstream
		savefile << filename << "I"; // append an I for 'Intensity' to this name to create the name of the output file
		ofstream myfile; // Create an outputstrem named myfile
		myfile.open(savefile.str()); // use open property of outputstream to create/open the *.debI file -> this name is achieved by using the string property of the stringstream
		for (int i = 0; i < nTT; i++) { // write output to myfile instead to the cout (CommandWindow Out)
			myfile << TwoTheta[i] << "\t" << Isum[i] << endl;
		}
		myfile.close(); // close the file
		time_end = clock();
		elapsed_time = float(time_end - time_start) / CLOCKS_PER_SEC;
		cout << "Time for Writing Output File: " << elapsed_time << " s" << endl;
	}
	return;
}